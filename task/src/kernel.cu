#include "hip/hip_runtime.h"
#include "defs.h"
#include "cudaErrorHandler.h"
#include "hip/hip_runtime.h"

#include <cstdio>

#define get_elem( _pptr_, _row_, _col_ ) ( *( ( MATRIX_TYPE* )( ( char* )_pptr_.ptr + _row_ * _pptr_.pitch ) + _col_ ) )

//---------------------------------------------------------------

__global__ void sum_kernel( hipPitchedPtr aDev, hipPitchedPtr bDev, hipPitchedPtr cDev, int aH, int aW, int bW )
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if( row > aH || col > bW || row < 0 || col < 0 ) 
        return;

    MATRIX_TYPE temp = MATRIX_TYPE(0);
    for ( int i = 0; i < aW; ++i )
        temp += get_elem( aDev, row, i ) * get_elem( bDev, i, col );

    get_elem( cDev, row, col ) = temp;
}

//---------------------------------------------------------------

void launchKernel( hipPitchedPtr aDev, hipPitchedPtr bDev, hipPitchedPtr cDev, int aH, int aW, int bW )
{
   dim3 block = dim3 ( BLOCK_SIZE, BLOCK_SIZE );
   dim3 grid = dim3 ( aH / BLOCK_SIZE, bW / BLOCK_SIZE );
   SAFE_KERNEL_CALL( ( sum_kernel<<< grid, block >>>( aDev, bDev, cDev, aH, aW, bW ) ) );
}

//---------------------------------------------------------------
